
#include <hip/hip_runtime.h>
#include <stdio.h>

#define SIZE	1024

__global__ void VectorAdd(int *a, int *b, int *c, int n)
{
	int i = threadIdx.x;

	if (i < n)
		c[i] = a[i] + b[i];
}

int main()
{
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;

	a = (int *)malloc(SIZE*sizeof(int));
	b = (int *)malloc(SIZE*sizeof(int));
	c = (int *)malloc(SIZE*sizeof(int));

	hipMalloc( &d_a, SIZE*sizeof(int));
	hipMalloc( &d_b, SIZE*sizeof(int));
	hipMalloc( &d_c, SIZE*sizeof(int));

	for( int i = 0; i < SIZE; ++i )
	{
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}

	hipMemcpy( d_a, a, SIZE*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, SIZE*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( d_c, c, SIZE*sizeof(int), hipMemcpyHostToDevice );

	// blocks, threads
	VectorAdd<<< 1, SIZE >>>(d_a, d_b, d_c, SIZE);
	
	hipMemcpy( c, d_c, SIZE*sizeof(int), hipMemcpyDeviceToHost );

	for( int i = 0; i < 10; ++i)
		printf("c[%d] = %d\n", i, c[i]);

	free(a);
	free(b);
	free(c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
